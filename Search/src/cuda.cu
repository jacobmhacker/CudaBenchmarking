
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void max_val(int* d_max, int* arr, int n) {
	int base = threadIdx.x * n;
	int max = *(arr + base);

	printf("In thread %d\n", threadIdx.x);

	for(int i = base + 1; i < base + n; i++) {
		if(*(arr + i) > max) {
			max = *(arr + i);
		}
	}

	*(d_max + threadIdx.x) = max;
}

/*
 * stdin takes the following inputs:
 *  - #threads
 *  - Array size
 *  - Array
 */

int main() {
	int n;
	int* arr;
	int* max;
	int* d_arr;
	int* d_max;

	int threads;

	scanf("%d", &threads);
	scanf("%d", &n);
	arr = (int*) malloc(sizeof(int) * n);
	max = (int*) malloc(sizeof(int) * threads);

	for(int i = 0; i < n; i++) {
		scanf("%d", arr + i);
	}

	hipMalloc((void**) &d_arr, sizeof(int) * n);
	hipMalloc((void**) &d_max, sizeof(int) * threads);
	hipMemcpy(d_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice);

	max_val<<<1,threads>>>(d_max, d_arr, n / threads);

	hipMemcpy(max, d_max, sizeof(int) * threads, hipMemcpyDeviceToHost);

	int max_val = *(max);

	for(int i = 1; i < threads; i++) {
		if(max_val < *(max + i)) {
			max_val = *(max + i);
		}
	}

	printf("Maximum value is: %d\n", max_val);

	return 0;
}
